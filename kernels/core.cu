#include "core.cuh"
#include "internal.cuh"

#include <string>

namespace kernels {

template<class T, int dim>
NDBuffer<T, dim>::NDBuffer(const std::array<size_t, dim>& sizes, T* const copy_from)
        : m_sizes(sizes)
        , m_size(std::accumulate(sizes.begin(), sizes.end(), 1, std::multiplies<size_t>()))
        , dev_ptr(NDBuffer::allocate(copy_from))
{}

template<class T, int dim>
T* NDBuffer<T, dim>::allocate(T* const copy_from) {
    const auto buffer_size = m_size * sizeof(T);
    CHECK_CUDA_CODE(hipMalloc(&dev_ptr, buffer_size));
    if (copy_from) {
        CHECK_CUDA_CODE(hipMemcpy(dev_ptr, copy_from, buffer_size, hipMemcpyHostToDevice));
    }
    return dev_ptr;
}

template<class T, int dim>
void NDBuffer<T, dim>::copy_to_host(T* host_ptr, int size) {
    CHECK_CUDA_CODE(hipMemcpy((void*)host_ptr, this->vptr(), size * sizeof(T), hipMemcpyDeviceToHost));
}

template<class T, int dim>
NDBuffer<T, dim>::~NDBuffer() {
    CHECK_CUDA_CODE(hipFree(dev_ptr));
}

template class NDBuffer<float, 1>;
template class NDBuffer<float, 2>;

}

