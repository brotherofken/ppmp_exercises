#include "core.cuh"
#include "internal.cuh"

#include <string>

namespace kernels {

template<class T, int dim>
NDBuffer<T, dim>::NDBuffer(const std::array<size_t, dim>& sizes, T* const copy_from)
        : m_sizes(sizes)
        , m_size(std::accumulate(sizes.begin(), sizes.end(), 1, std::multiplies<size_t>()))
        , dev_ptr(NDBuffer::allocate(copy_from))
{}

template<class T, int dim>
T* NDBuffer<T, dim>::allocate(T* const copy_from) {
    const auto buffer_size = m_size * sizeof(T);
    CHECK_CUDA_CODE(hipMalloc(&dev_ptr, buffer_size));
    if (copy_from) {
        copy_to_device(copy_from, m_size);
    }
    return dev_ptr;
}

template<class T, int dim>
void NDBuffer<T, dim>::copy_to_host(T* host_ptr, int size) {
    CHECK_CUDA_CODE(hipMemcpy((void*)host_ptr, this->vptr(), size * sizeof(T), hipMemcpyDeviceToHost));
}

template<class T, int dim>
void NDBuffer<T, dim>::copy_to_device(T* host_ptr, int size) {
    CHECK_CUDA_CODE(hipMemcpy(this->vptr(), (void*)host_ptr, size * sizeof(T), hipMemcpyHostToDevice));
}

template<class T, int dim>
NDBuffer<T, dim>::~NDBuffer() {
    CHECK_CUDA_CODE(hipFree(dev_ptr));
}

struct CudaStopwatchData {
    hipEvent_t event_start;
    hipEvent_t event_stop;
};

CudaStopwatch::CudaStopwatch()
    : data(std::make_shared<CudaStopwatchData>())
{
    hipEventCreate(&data->event_start);
    hipEventCreate(&data->event_stop);
}

void CudaStopwatch::start() {
    hipEventRecord(data->event_start);
}

void CudaStopwatch::stop() {
    hipEventRecord(data->event_stop);
}

float CudaStopwatch::elapsedMs() {
    float milliseconds = 0;
    hipEventSynchronize(data->event_stop);
    hipEventElapsedTime(&milliseconds, data->event_start, data->event_stop);
    return milliseconds;
}

float CudaStopwatch::elapsedS() {
    return elapsedMs() / 1000.f;
}

template class NDBuffer<float, 1>;
template class NDBuffer<float, 2>;

}
