#include "core.cuh"
#include "internal.cuh"

#include <string>

namespace kernels {

template<class T, int dim>
NDSpan<T, dim>::NDSpan(const std::array<size_t, dim>& sizes, T* const copy_from)
        : m_sizes(sizes)
        , m_size(std::accumulate(sizes.begin(), sizes.end(), 1, std::multiplies<size_t>()))
        , dev_ptr(NDSpan::allocate(copy_from))
{}

template<class T, int dim>
T* NDSpan<T, dim>::allocate(T* const copy_from) {
    const auto buffer_size = m_size * sizeof(T);
    CHECK_CUDA_CODE(hipMalloc(&dev_ptr, buffer_size));
    if (copy_from) {
        CHECK_CUDA_CODE(hipMemcpy(dev_ptr, copy_from, buffer_size, hipMemcpyHostToDevice));
    }
    return dev_ptr;
}

template<class T, int dim>
void NDSpan<T, dim>::copy_to_host(T* host_ptr, int size) {
    CHECK_CUDA_CODE(hipMemcpy((void*)host_ptr, this->vptr(), size * sizeof(T), hipMemcpyDeviceToHost));
}

template<class T, int dim>
NDSpan<T, dim>::~NDSpan() {
    CHECK_CUDA_CODE(hipFree(dev_ptr));
}

template class NDSpan<float, 1>;
template class NDSpan<float, 2>;

}

