#include "device_query.cuh"

#include <fmt/format.h>

namespace kernels {

__host__ void device_query() {
    const int deviceCount = []{
        int result = 0;
        hipGetDeviceCount(&result);
        return result;
    }();

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp{};
        hipGetDeviceProperties(&deviceProp, dev);

        if (dev == 0) {
            if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
                fmt::print("No CUDA GPU has been detected\n");
                return;
            } else if (deviceCount == 1) {
                fmt::print("There is 1 device supporting CUDA\n");
            } else {
                fmt::print("There are {} devices supporting CUDA\n", deviceCount);
            }
        }

        fmt::print("Device {} name {}", dev, deviceProp.name);
        fmt::print("\tComputational Capabilities: {}.{}\n", deviceProp.major, deviceProp.minor);
        fmt::print("\tMaximum global memory msize: {}\n", deviceProp.totalGlobalMem);
        fmt::print("\tMaximum constant memory msize: {}\n", deviceProp.totalConstMem);
        fmt::print("\tMaximum shared memory msize per block: {}\n", deviceProp.sharedMemPerBlock);
        fmt::print("\tMaximum block dimensions: [{}]\n", fmt::join(deviceProp.maxThreadsDim, " x "));
        fmt::print("\tMaximum grid dimensions: [{}]\n", fmt::join(deviceProp.maxGridSize, " x "));
        fmt::print("\tWarp msize: {}\n", deviceProp.warpSize);
    }
}

}
