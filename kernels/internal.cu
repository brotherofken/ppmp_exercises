#include "internal.cuh"

#include <stdexcept>
#include <string>

#include <fmt/format.h>

namespace kernels {

    void check_cuda_error_code(hipError_t code, const char* file, int line) {
        if (code != hipSuccess) {
            const auto msg = fmt::format("Something weird happened at {}:{}. Error {}, message: {}\n",
                                         std::string(file),
                                         line,
                                         hipGetErrorName(code),
                                         hipGetErrorString(code)
            );
            fmt::print(msg);
            throw std::runtime_error(msg);
        }
    }

}